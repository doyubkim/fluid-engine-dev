#include "hip/hip_runtime.h"
// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <jet/constants.h>
#include <jet/cuda_particle_system_solver3.h>
#include <jet/timer.h>

#include <benchmark/benchmark.h>

#include <thrust/device_ptr.h>
#include <thrust/random.h>
#include <thrust/transform.h>

namespace {

struct Rng {
    template <typename Index>
    __device__ float4 operator()(Index idx) {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist(0.0f, 1.0f);

        float4 result;
        randEng.discard(3 * idx);
        result.x = uniDist(randEng);
        randEng.discard(3 * idx + 1);
        result.y = uniDist(randEng);
        randEng.discard(3 * idx + 2);
        result.z = uniDist(randEng);
        result.w = 0.0f;

        return result;
    }
};

}  // namespace

class CudaParticleSystemSolver3 : public benchmark::Fixture {
 public:
    jet::CudaParticleSystemSolver3 solver;
    jet::Frame frame{0, 1.0 / 300.0};

    void SetUp(benchmark::State& state) override {
        solver.setDragCoefficient(0.0);
        solver.setRestitutionCoefficient(1.0);

        size_t numParticles = static_cast<size_t>(state.range(0));
        auto particles = solver.particleSystemData();

        jet::CudaArray1<float4> pos(numParticles);
        thrust::transform(thrust::make_counting_iterator(jet::kZeroSize),
            thrust::make_counting_iterator(numParticles),
            thrust::device_ptr<float4>(pos.data()),
            Rng());
        particles->addParticles(jet::CudaArrayView1<float4>(pos));
    }

    void SetUp(const benchmark::State&) override {}

    void TearDown(benchmark::State&) override {
        solver = jet::CudaParticleSystemSolver3();
    }

    void TearDown(const benchmark::State&) override {}

    void update() {
        solver.update(frame);
        frame.advance();
        hipDeviceSynchronize();
    }
};

BENCHMARK_DEFINE_F(CudaParticleSystemSolver3, Update)
(benchmark::State& state) {
    using namespace std::chrono;

    while (state.KeepRunning()) {
        jet::Timer timer;

        update();

        const double elapsedSeconds = timer.durationInSeconds();

        state.SetIterationTime(elapsedSeconds);
    }
}
BENCHMARK_REGISTER_F(CudaParticleSystemSolver3, Update)
    ->Arg(1 << 18)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond);
