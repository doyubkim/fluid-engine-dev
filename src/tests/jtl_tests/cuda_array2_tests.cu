// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <jet/cuda_array.h>
#include <jet/cuda_array_view.h>

#include <gtest/gtest.h>

using namespace jet;

TEST(CudaArray2, Constructors) {
    {
        CudaArray2<float> arr;
        EXPECT_EQ(0u, arr.width());
        EXPECT_EQ(0u, arr.height());
    }
    {
        CudaArray2<float> arr(CudaStdArray<size_t, 2>(3, 7));
        EXPECT_EQ(3u, arr.width());
        EXPECT_EQ(7u, arr.height());
        for (size_t i = 0; i < 21; ++i) {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }
    }
    {
        CudaArray2<float> arr(CudaStdArray<size_t, 2>(1, 9), 1.5f);
        EXPECT_EQ(1u, arr.width());
        EXPECT_EQ(9u, arr.height());
        for (size_t i = 0; i < 9; ++i) {
            EXPECT_FLOAT_EQ(1.5f, arr[i]);
        }
    }
    {
        CudaArray2<float> arr(5, 2);
        EXPECT_EQ(5u, arr.width());
        EXPECT_EQ(2u, arr.height());
        for (size_t i = 0; i < 10; ++i) {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }
    }
    {
        CudaArray2<float> arr(3, 4, 7.f);
        EXPECT_EQ(3u, arr.width());
        EXPECT_EQ(4u, arr.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ(7.f, arr[i]);
        }
    }
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        EXPECT_EQ(4u, arr.width());
        EXPECT_EQ(3u, arr.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr[i]);
        }
    }
    {
        Array2<float> arr({{1.f, 2.f, 3.f, 4.f},
                           {5.f, 6.f, 7.f, 8.f},
                           {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(arr);
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(arr);
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        CudaArrayView2<float> arrVew(arr.data(), arr.size());
        EXPECT_EQ(4u, arrVew.width());
        EXPECT_EQ(3u, arrVew.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arrVew[i]);
        }
    }
}

TEST(CudaArray2, At) {
    {
        float values[12] = {0.f, 1.f, 2.f, 3.f, 4.f,  5.f,
                            6.f, 7.f, 8.f, 9.f, 10.f, 11.f};
        CudaArray2<float> arr(4, 3);
        for (size_t i = 0; i < 12; ++i) {
            arr[i] = values[i];
        }

        // Test row-major
        EXPECT_FLOAT_EQ(0.f, arr(0, 0));
        EXPECT_FLOAT_EQ(1.f, arr(1, 0));
        EXPECT_FLOAT_EQ(2.f, arr(2, 0));
        EXPECT_FLOAT_EQ(3.f, arr(3, 0));
        EXPECT_FLOAT_EQ(4.f, arr(0, 1));
        EXPECT_FLOAT_EQ(5.f, arr(1, 1));
        EXPECT_FLOAT_EQ(6.f, arr(2, 1));
        EXPECT_FLOAT_EQ(7.f, arr(3, 1));
        EXPECT_FLOAT_EQ(8.f, arr(0, 2));
        EXPECT_FLOAT_EQ(9.f, arr(1, 2));
        EXPECT_FLOAT_EQ(10.f, arr(2, 2));
        EXPECT_FLOAT_EQ(11.f, arr(3, 2));
    }
}

TEST(CudaArray2, CopyFrom) {
    // From Array
    {
        Array2<float> arr({{1.f, 2.f, 3.f, 4.f},
                           {5.f, 6.f, 7.f, 8.f},
                           {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(2, 5);

        arr2.copyFrom(arr);
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From ArrayView
    {
        Array2<float> arr({{1.f, 2.f, 3.f, 4.f},
                           {5.f, 6.f, 7.f, 8.f},
                           {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(2, 5);

        arr2.copyFrom(arr.view());
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CudaArray
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(2, 5);

        arr2.copyFrom(arr);
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CudaArrayView
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(2, 5);

        arr2.copyFrom(arr.view());
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
}

TEST(CudaArray2, CopyTo) {
    // To Array
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        Array2<float> arr2(2, 5);

        arr.copyTo(arr2);
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // To ArrayView
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        Array2<float> arr2(4, 3);

        arr.copyTo(arr2.view());
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CudaArray
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(2, 5);

        arr.copyTo(arr2);
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CudaArrayView
    {
        CudaArray2<float> arr({{1.f, 2.f, 3.f, 4.f},
                               {5.f, 6.f, 7.f, 8.f},
                               {9.f, 10.f, 11.f, 12.f}});
        CudaArray2<float> arr2(4, 3);

        arr.copyTo(arr2.view());
        EXPECT_EQ(4u, arr2.width());
        EXPECT_EQ(3u, arr2.height());
        for (size_t i = 0; i < 12; ++i) {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
}

TEST(CudaArray2, Fill) {
    CudaArray2<float> arr(
        {{1.f, 2.f, 3.f, 4.f}, {5.f, 6.f, 7.f, 8.f}, {9.f, 10.f, 11.f, 12.f}});

    arr.fill(42.0f);
    EXPECT_EQ(4u, arr.width());
    EXPECT_EQ(3u, arr.height());
    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ(42.0f, arr[i]);
    }
}

TEST(CudaArray2, Resize) {
    {
        CudaArray2<float> arr;
        arr.resize(CudaStdArray<size_t, 2>(2, 9));
        EXPECT_EQ(2u, arr.width());
        EXPECT_EQ(9u, arr.height());
        for (size_t i = 0; i < 18; ++i) {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }

        arr.resize(CudaStdArray<size_t, 2>(8, 13), 4.f);
        hipDeviceSynchronize();
        EXPECT_EQ(8u, arr.width());
        EXPECT_EQ(13u, arr.height());
        for (size_t i = 0; i < 8; ++i) {
            for (size_t j = 0; j < 13; ++j) {
                if (i < 2 && j < 9) {
                    EXPECT_FLOAT_EQ(0.f, arr(i, j));
                } else {
                    EXPECT_FLOAT_EQ(4.f, arr(i, j));
                }
            }
        }
    }
    {
        CudaArray2<float> arr;
        arr.resize(7, 6);
        EXPECT_EQ(7u, arr.width());
        EXPECT_EQ(6u, arr.height());
        for (size_t i = 0; i < 42; ++i) {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }

        arr.resize(1, 9, 3.f);
        EXPECT_EQ(1u, arr.width());
        EXPECT_EQ(9u, arr.height());
        for (size_t i = 0; i < 1; ++i) {
            for (size_t j = 0; j < 9; ++j) {
                if (j < 6) {
                    EXPECT_FLOAT_EQ(0.f, arr(i, j));
                } else {
                    EXPECT_FLOAT_EQ(3.f, arr(i, j));
                }
            }
        }
    }
}

TEST(CudaArray2, Clear) {
    CudaArray2<float> arr(
        {{1.f, 2.f, 3.f, 4.f}, {5.f, 6.f, 7.f, 8.f}, {9.f, 10.f, 11.f, 12.f}});

    arr.clear();
    EXPECT_EQ(0u, arr.width());
    EXPECT_EQ(0u, arr.height());
}

TEST(CudaArray2, Swap) {
    CudaArray2<float> arr(
        {{1.f, 2.f, 3.f, 4.f}, {5.f, 6.f, 7.f, 8.f}, {9.f, 10.f, 11.f, 12.f}});
    CudaArray2<float> arr2(2, 5, 42.f);

    arr.swap(arr2);

    EXPECT_EQ(2u, arr.width());
    EXPECT_EQ(5u, arr.height());
    for (size_t i = 0; i < 10; ++i) {
        EXPECT_FLOAT_EQ(42.0f, arr[i]);
    }

    EXPECT_EQ(4u, arr2.width());
    EXPECT_EQ(3u, arr2.height());
    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
    }
}

TEST(CudaArray2, View) {
    CudaArray2<float> arr(
        {{1.f, 2.f, 3.f, 4.f}, {5.f, 6.f, 7.f, 8.f}, {9.f, 10.f, 11.f, 12.f}});

    auto view = arr.view();

    EXPECT_EQ(4u, view.width());
    EXPECT_EQ(3u, view.height());
    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ((float)i + 1.f, view[i]);
    }
    for (size_t j = 0; j < 3; ++j) {
        for (size_t i = 0; i < 4; ++i) {
            EXPECT_FLOAT_EQ(arr(i, j), view(i, j));
        }
    }

    const auto& arrRef = arr;
    auto constView = arrRef.view();

    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ((float)i + 1.f, constView[i]);
    }
    for (size_t j = 0; j < 3; ++j) {
        for (size_t i = 0; i < 4; ++i) {
            EXPECT_FLOAT_EQ(arr(i, j), constView(i, j));
        }
    }

    for (size_t j = 0; j < 3; ++j) {
        for (size_t i = 0; i < 4; ++i) {
            view(i, j) = float(i + 4 * j);
        }
    }

    for (size_t j = 0; j < 3; ++j) {
        for (size_t i = 0; i < 4; ++i) {
            EXPECT_FLOAT_EQ(float(i + 4 * j), arr(i, j));
            EXPECT_FLOAT_EQ(float(i + 4 * j), constView(i, j));
        }
    }
}

TEST(CudaArray2, AssignmentOperator) {
    CudaArray2<float> arr(
        {{1.f, 2.f, 3.f, 4.f}, {5.f, 6.f, 7.f, 8.f}, {9.f, 10.f, 11.f, 12.f}});
    CudaArray2<float> arr2(2, 5, 42.f);

    arr2 = arr;

    EXPECT_EQ(4u, arr.width());
    EXPECT_EQ(3u, arr.height());
    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr[i]);
    }

    EXPECT_EQ(4u, arr2.width());
    EXPECT_EQ(3u, arr2.height());
    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
    }
}

TEST(CudaArray2, MoveOperator) {
    CudaArray2<float> arr(
        {{1.f, 2.f, 3.f, 4.f}, {5.f, 6.f, 7.f, 8.f}, {9.f, 10.f, 11.f, 12.f}});
    CudaArray2<float> arr2(2, 5, 42.f);

    arr2 = std::move(arr);

    EXPECT_EQ(0u, arr.width());
    EXPECT_EQ(0u, arr.height());
    EXPECT_EQ(nullptr, arr.data());
    EXPECT_EQ(4u, arr2.width());
    EXPECT_EQ(3u, arr2.height());
    for (size_t i = 0; i < 12; ++i) {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
    }
}
