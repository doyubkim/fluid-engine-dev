#include "hip/hip_runtime.h"
// Copyright (c) 2018 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <pch.h>

#include <jet/constants.h>
#include <jet/cuda_sph_kernels3.h>
#include <jet/cuda_utils.h>
#include <jet/cuda_wc_sph_solver3.h>
#include <jet/timer.h>

using namespace jet;

namespace {

inline __device__ float computePressureFromEos(float density,
                                               float targetDensity,
                                               float eosScale,
                                               float eosExponent,
                                               float negativePressureScale) {
    // Equation of state
    // (http://www.ifi.uzh.ch/vmml/publications/pcisph/pcisph.pdf)
    float p = eosScale / eosExponent *
              (powf((density / targetDensity), eosExponent) - 1.0f);

    // Negative pressure scaling
    if (p < 0) {
        p *= negativePressureScale;
    }

    return p;
}

__global__ void computePressureKernel(float targetDensity, float eosScale,
                                      float eosExponent,
                                      float negativePressureScale,
                                      const float* densities, size_t n,
                                      float* pressures) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        pressures[i] =
            computePressureFromEos(densities[i], targetDensity, eosScale,
                                   eosExponent, negativePressureScale);
    }
}

__global__ void computeForcesKernel(
    float mass, float4 gravity, float viscosity,
    CudaSphSpikyKernel3 spikyKernel, const uint32_t* neighborStarts,
    const uint32_t* neighborEnds, const uint32_t* neighborLists,
    const float4* positions, const float4* velocities, const float* densities,
    const float* pressures, size_t n, float4* smoothedVelocities,
    float4* forces) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        uint32_t ns = neighborStarts[i];
        uint32_t ne = neighborEnds[i];

        float4 x_i = positions[i];
        float4 v_i = velocities[i];
        float d_i = densities[i];
        float p_i = pressures[i];
        float4 f = gravity;

        float massSquared = mass * mass;

        float w_i = mass / d_i * spikyKernel(0.0f);
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj) {
            uint32_t j = neighborLists[jj];

            float4 r = positions[j] - x_i;
            float dist = length(r);

            if (dist > 0.0f) {
                float4 dir = r / dist;

                float4 v_j = velocities[j];
                float d_j = densities[j];
                float p_j = pressures[j];

                // Pressure force
                f -= massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     spikyKernel.gradient(dist, dir);

                // Viscosity force
                f += viscosity * massSquared * (v_j - v_i) / d_j *
                     spikyKernel.secondDerivative(dist);

                // Pseudo viscosity
                float w_j = mass / d_j * spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        forces[i] = f;

        smoothedVelocity /= weightSum;
        smoothedVelocities[i] = smoothedVelocity;
    }
}

#define BND_R 0.0f

__global__ void timeIntegrationKernel(float dt, float mass, float smoothFactor,
                                      float3 lower, float3 upper,
                                      const float4* smoothedVelocities,
                                      const float4* forces, size_t n,
                                      float4* positions, float4* velocities) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float4 x = positions[i];
        float4 v = velocities[i];
        float4 s = smoothedVelocities[i];
        float4 f = forces[i];

        v = (1.0f - smoothFactor) * v + smoothFactor * s;
        v += dt * f / mass;
        x += dt * v;

        // TODO: Add proper collider support
        if (x.x > upper.x) {
            x.x = upper.x;
            v.x *= BND_R;
        }
        if (x.x < lower.x) {
            x.x = lower.x;
            v.x *= BND_R;
        }
        if (x.y > upper.y) {
            x.y = upper.y;
            v.y *= BND_R;
        }
        if (x.y < lower.y) {
            x.y = lower.y;
            v.y *= BND_R;
        }
        if (x.z > upper.z) {
            x.z = upper.z;
            v.z *= BND_R;
        }
        if (x.z < lower.z) {
            x.z = lower.z;
            v.z *= BND_R;
        }

        positions[i] = x;
        velocities[i] = v;
    }
}

}  // namespace

void CudaWcSphSolver3::onAdvanceTimeStep(double timeStepInSeconds) {
    auto sph = sphSystemData();
    size_t n = sph->numberOfParticles();

    // Build neighbor searcher
    sph->buildNeighborSearcher();
    sph->buildNeighborListsAndUpdateDensities();

    // Compute pressure
    auto d = sph->densities();
    auto p = sph->pressures();
    const float targetDensity = sph->targetDensity();
    const float eosScale =
        targetDensity * square(speedOfSound()) / _eosExponent;

    unsigned int numBlocks, numThreads;
    cudaComputeGridSize((unsigned int)n, 256, numBlocks, numThreads);

    computePressureKernel<<<numBlocks, numThreads>>>(
        targetDensity, eosScale, _eosExponent, negativePressureScale(),
        d.data(), n, p.data());

    // Compute pressure / viscosity forces and smoothed velocity
    float mass = sph->mass();
    float h = sph->kernelRadius();
    auto ns = sph->neighborStarts();
    auto ne = sph->neighborEnds();
    auto nl = sph->neighborLists();
    auto x = sph->positions();
    auto v = sph->velocities();
    auto s = smoothedVelocities();
    auto f = forces();

    computeForcesKernel<<<numBlocks, numThreads>>>(
        mass, toFloat4(gravity(), 0.0f), viscosityCoefficient(),
        CudaSphSpikyKernel3(h), ns.data(), ne.data(), nl.data(), x.data(),
        v.data(), d.data(), p.data(), n, s.data(), f.data());

    // Time-integration
    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * pseudoViscosityCoefficient();
    factor = clamp(factor, 0.0f, 1.0f);
    auto lower = toFloat3(container().lowerCorner);
    auto upper = toFloat3(container().upperCorner);

    timeIntegrationKernel<<<numBlocks, numThreads>>>(dt, mass, factor, lower,
                                                     upper, s.data(), f.data(),
                                                     n, x.data(), v.data());
}
